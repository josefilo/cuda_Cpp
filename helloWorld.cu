#include<stdio.h>
#include "hip/hip_runtime.h"
// kernel
__global__ void meuKernel(){
    
    printf("Hello World da GPU!\n");

    
}

//codigo de CPU

int main(){
    printf("Hello World!\n");
    meuKernel <<<2,2>>>();
    /*É preciso colocar para que a CPU espere a GPU terminar,
    caso não coloque, o programa termina antes que a GPU imprima os dados*/
    hipDeviceSynchronize();
    return 0;
}
